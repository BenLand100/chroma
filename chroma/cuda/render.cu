#include "hip/hip_runtime.h"
//-*-c-*-

#include "linalg.h"
#include "intersect.h"
#include "mesh.h"
#include "sorting.h"
#include "geometry.h"

#include "stdio.h"

__device__ float4
get_color(const float3 &direction, const Triangle &t, unsigned int rgba)
{
    float3 v01 = t.v1 - t.v0;
    float3 v12 = t.v2 - t.v1;
    
    float3 surface_normal = normalize(cross(v01,v12));

    float cos_theta = dot(surface_normal,-direction);

    if (cos_theta < 0.0f)
	cos_theta = -cos_theta;

    unsigned int a0 = 0xff & (rgba >> 24);
    unsigned int r0 = 0xff & (rgba >> 16);
    unsigned int g0 = 0xff & (rgba >> 8);
    unsigned int b0 = 0xff & rgba;

    float alpha = (255 - a0)/255.0f;

    return make_float4(r0*cos_theta, g0*cos_theta, b0*cos_theta, alpha);
}

extern "C"
{

__global__ void
render(int nthreads, float3 *_origin, float3 *_direction, Geometry *g,
       unsigned int alpha_depth, unsigned int *pixels, float *_dx,
       unsigned int *dxlen, float4 *_color)
{
    __shared__ Geometry sg;

    if (threadIdx.x == 0)
	sg = *g;

    __syncthreads();

    int id = blockIdx.x*blockDim.x + threadIdx.x;
	
    if (id >= nthreads)
	return;

    g = &sg;

    float3 origin = _origin[id];
    float3 direction = _direction[id];
    unsigned int n = dxlen[id];

    float distance;

    if (n < 1 && !intersect_node(origin, direction, g, g->start_node)) {
	pixels[id] = 0;
	return;
    }

    unsigned int stack[STACK_SIZE];

    unsigned int *head = &stack[0];
    unsigned int *node = &stack[1];
    unsigned int *tail = &stack[STACK_SIZE-1];
    *node = g->start_node;

    float *dx = _dx + id*alpha_depth;
    float4 *color_a = _color + id*alpha_depth;

    unsigned int i;

    do {
	unsigned int first_child = g->node_map[*node];
	unsigned int stop = g->node_map_end[*node];
	
	while (*node >= g->first_node && stop == first_child+1) {
	    *node = first_child;
	    first_child = g->node_map[*node];
	    stop = g->node_map_end[*node];
	}
		
	if (*node >= g->first_node) {
	    for (i=first_child; i < stop; i++) {
		if (intersect_node(origin, direction, g, i)) {
		    *node = i;
		    node++;
		}
	    }

	    node--;
	}
	else {
	    // node is a leaf
	    for (i=first_child; i < stop; i++) {
		Triangle t = get_triangle(g, i);
		
		if (intersect_triangle(origin, direction, t, distance)) {
		    if (n < 1) {
			dx[0] = distance;
			    
			unsigned int rgba = g->colors[i];
			float4 color = get_color(direction, t, rgba);

			color_a[0] = color;
		    }
		    else {
			unsigned long j = searchsorted(n, dx, distance);

			if (j <= alpha_depth-1) {
			    insert(alpha_depth, dx, j, distance);

			    unsigned int rgba = g->colors[i];
			    float4 color = get_color(direction, t, rgba);

			    insert(alpha_depth, color_a, j, color);
			}
		    }
					
		    if (n < alpha_depth)
			n++;
		}
				
	    } // triangle loop
	    
	    node--;
	    
	} // node is a leaf
	
    } // while loop
    while (node != head);

    if (n < 1) {
	pixels[id] = 0;
	return;
    }

    dxlen[id] = n;

    float scale = 1.0f;
    float fr = 0.0f;
    float fg = 0.0f;
    float fb = 0.0f;
    for (i=0; i < n; i++) {
	float alpha = color_a[i].w;

	fr += scale*color_a[i].x*alpha;
	fg += scale*color_a[i].y*alpha;
	fb += scale*color_a[i].z*alpha;
	
	scale *= (1.0f-alpha);
    }
    unsigned int a;
    if (n < alpha_depth)
	a = floorf(255*(1.0f-scale));
    else
    	a = 255;
    unsigned int red = floorf(fr/(1.0f-scale));
    unsigned int green = floorf(fg/(1.0f-scale));
    unsigned int blue = floorf(fb/(1.0f-scale));

    pixels[id] = a << 24 | red << 16 | green << 8 | blue;
}

} // extern "C"
